#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
#define BlockSize 1024
 
using namespace std;


ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/

// in this function nodes are getting updated which are going to be in next level that is L+1 simultaneously gmax is getting set to the last node of that level 
//  so that we will get to know how much nodes are present in next level of the graph. 
__global__ void forth(int *csr_offset, int *csr_List, int *apr, int *aid, int *num_active, int *active, int *levels, int V, int E, int L,int l,int *gmax,int *gmin)
{
       int tid = threadIdx.x + blockIdx.x * blockDim.x+gmin[0];
    if(tid < V && levels[tid]==l)
        {   int from=csr_offset[tid],to= csr_offset[tid+1];
            int j=from;
            while(j<to)
            {
                levels[csr_List[j]] = l+1;
                atomicMax(&gmax[0], csr_List[j]);
                j++;
            }
        }
}

//in this function indegree of the the nodes present in level L+1 are incremented by the active nodes present in the level L. 
__global__ void third(int *csr_offset, int *csr_List, int *apr, int *aid, int *num_active, int *active, int *levels, int V, int E, int L,int l,int *gmax,int *gmin)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x+gmin[0];
    if(tid < V && levels[tid]==l && active[tid]==1)
    {
            int from=csr_offset[tid],to= csr_offset[tid+1];
            int j=from;
            while(j<to)
            {
                atomicAdd(&aid[csr_List[j]], 1);
                j++;
            }
    }
     
}


// in this kernel nodes which are active but there neighbours(left and right ones) are deactive and all three of them present in same level then then middle active node will
// get deactive
__global__ void second(int *csr_offset, int *csr_List, int *apr, int *aid, int *num_active, int *active, int *levels, int V, int E, int L,int l,int *gmax,int *gmin)
{
        int tid =threadIdx.x + blockIdx.x * blockDim.x+gmin[0];
       if(tid < V-1 && tid > 0 &&  active[tid] == 1 && levels[tid] == l)
         if( levels[tid-1] == l && active[tid-1] == 0 &&  levels[tid+1] == l &&  active[tid+1] == 0) 
                 {
                    atomicExch(&active[tid], 0);
                    atomicSub(&num_active[l], 1);                      
                 }

}


// in this function nodes whose activation point requirement is equal to the active in degree are getting updated  
__global__ void first(int *csr_offset, int *csr_List, int *apr, int *aid, int *num_active, int *active, int *levels, int V, int E, int L,int l,int *gmax,int *gmin)
{
        int tid = threadIdx.x + blockIdx.x * blockDim.x + gmin[0];
        if(tid < V && levels[tid] == l && aid[tid] >= apr[tid] && active[tid]==0)
        {
            atomicExch(&active[tid], 1);
            atomicAdd(&num_active[l], 1);
        }
        
}


// in this fucntion nodes of level 0 are getting updated and simultaneously we will set gmax equal to the number of nodes in level 0 .  
__global__ void ini_kernel( int *apr ,int *num_active, int *active, int *levels, int V,int *gmax)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < V && apr[tid] == 0)
    {    
        active[tid]= 1;
        levels[tid]= 0;
        atomicAdd(&num_active[0], 1);
        atomicMax(&gmax[0], tid);
    }
}
    
    
/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    int *d_activeVertex;
	  hipMalloc(&d_activeVertex, L*sizeof(int));


/***Important***/

// Initialize d_aid array to zero for each vertex
// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/
hipMemset(d_aid, 0, V*sizeof(int));
hipMemcpy(d_activeVertex, h_activeVertex, L*sizeof(int), hipMemcpyHostToDevice);
int *d_active;
hipMalloc(&d_active, V*sizeof(int));
hipMemset(d_active, 0, V*sizeof(int));
int *d_levels;
hipMalloc(&d_levels, V*sizeof(int));
hipMemset(d_levels, -1, V*sizeof(int));
int grid_size = ceil((float)V / 1024);


int cmax[1],*gmax,cmin[1],*gmin;
cmax[0]=0;
cmin[0]=0;
// cmax is the last node of that level and cmin is the first node of that level in cpu.
// gmax is the last node of that level and gmin is the first node of that level in gpu.
hipMalloc(&gmax,sizeof(int));
hipMalloc(&gmin,sizeof(int));
hipMemcpy(gmax , cmax, sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(gmin , cmin, sizeof(int), hipMemcpyHostToDevice);

//( int *apr ,int *num_active, int *active, int *levels, int V,int *gmax)
ini_kernel<<<grid_size,1024>>>(d_apr,d_activeVertex,d_active,d_levels,V,gmax);
hipMemcpy(cmax , gmax, sizeof(int), hipMemcpyDeviceToHost);
printf("%d",cmax[0]);
int l=0;
    while(l<L)
    {  

       grid_size= ceil((float)(cmax[0]-cmin[0]+1)/1024);
      first<<<grid_size,1024>>>(d_offset,d_csrList,d_apr,d_aid,d_activeVertex,d_active,d_levels,V,E,L,l,gmax,gmin);     
      second<<<grid_size,1024>>>(d_offset,d_csrList,d_apr,d_aid,d_activeVertex,d_active,d_levels,V,E,L,l,gmax,gmin);
      third<<<grid_size,1024>>>(d_offset,d_csrList,d_apr,d_aid,d_activeVertex,d_active,d_levels,V,E,L,l,gmax,gmin);
      forth<<<grid_size,1024>>>(d_offset,d_csrList,d_apr,d_aid,d_activeVertex,d_active,d_levels,V,E,L,l,gmax,gmin);
      cmin[0]=cmax[0];
      hipMemcpy(cmax , gmax, sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(gmin , cmin, sizeof(int), hipMemcpyHostToDevice);

      l++;
    }
hipMemcpy(h_activeVertex, d_activeVertex, L*sizeof(int), hipMemcpyDeviceToHost);
/********************************END OF CODE AREA**********************************/
double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}
